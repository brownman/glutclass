#include "hip/hip_runtime.h"
#include </usr/local/cuda/include/hip/hip_runtime.h>
#include "../glclasses/glut.h"
#include "../glclasses/pixmap.h"


// ***************************  CUDA kernel ***********************************************
__global__ void cuda_kernel( int scaleX, int scaleY,
                             int width, int height,
                             int stride, GLubyte *devPixmap ) {
    // first load up the bounds, and make sure we are within it
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    if (x >= width) return;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (y >= height) return;

    // then fill in the pixel values
    devPixmap[(stride * y + x)*4] = 0xFF;
    devPixmap[(stride * y + x)*4 +1] = (y - scaleY) * (x - scaleX);
    devPixmap[(stride * y + x)*4 +2] = 0;
    devPixmap[(stride * y + x)*4 +3] = 0xFF;
}

// ***************************  MyGlut class ***********************************************
class MyGlut : public Glut {
  public:
    MyGlut() : scaleX(0), scaleY(0) {}
    ~MyGlut() {
        hipFree( devPixmap_ );
    }

    virtual void init( int *argc, char **argv, int width, int height ) {
        Glut::init( argc, argv, width, height );
        pixmap_.init( width, height );
        hipMalloc( (void**)&devPixmap_, pixmap_.bytes() );
    }

    virtual void display_event( void ) {
        glDrawPixels( pixmap_.width(), pixmap_.height(), GL_RGBA, GL_UNSIGNED_BYTE, pixmap_.pixels() );
    }

    virtual void mouse_event( int button, int state, int x, int y ) {
        if (button == GLUT_LEFT_BUTTON && state == GLUT_DOWN) {
            lastX = x;
            lastY = y;
        }
    }

    virtual void mouse_active_motion_event( int x, int y ) {
        int deltaX = x - lastX;
        int deltaY = y - lastY;

        scaleX += deltaX;
        scaleY += deltaY;
        do_pixels();

        lastX = x;
        lastY = y;

        glutPostRedisplay();
    }

    virtual void reshape_event( int width, int height ) {
        pixmap_.resize( width, height, CLEAR | COPY );
        hipFree( (void*)devPixmap_ );
        hipMalloc( (void**)&devPixmap_, pixmap_.bytes() );
        do_pixels();
        Glut::reshape_event( width, height );
    }

    void do_pixels( void ) {
        dim3    grids((pixmap_.width()+15)/16,(pixmap_.height()+15)/16);
        dim3    threads(16,16);
        cuda_kernel<<<grids,threads>>>( scaleX, scaleY,
                                        pixmap_.width(), pixmap_.height(),
                                        pixmap_.stride(), devPixmap_ );
        hipMemcpy( pixmap_.pixels(), devPixmap_, pixmap_.bytes(), hipMemcpyDeviceToHost );
    }

  private:
    Pixmap  pixmap_;
    GLubyte *devPixmap_;
    int     scaleX, scaleY;
    int     lastX, lastY; // used for moving mouse
};

int main( int argc, char **argv ) {
    MyGlut  glut;

    glut.init( &argc, argv, 1024, 768 );
    glut.do_pixels();

    int     exitCode = 0;
    try {
        glut.loop();
    }
    catch (int e) {
        exitCode = e;
    }

    return exitCode;
}
